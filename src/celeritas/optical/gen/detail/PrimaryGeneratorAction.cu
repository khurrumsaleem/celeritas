#include "hip/hip_runtime.h"
//------------------------------ -*- cuda -*- -------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/optical/gen/detail/PrimaryGeneratorAction.cu
//---------------------------------------------------------------------------//
#include "PrimaryGeneratorAction.hh"

#include "corecel/Assert.hh"
#include "corecel/sys/KernelLauncher.device.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/optical/CoreParams.hh"
#include "celeritas/optical/CoreState.hh"
#include "celeritas/optical/MaterialParams.hh"
#include "celeritas/optical/action/ActionLauncher.device.hh"
#include "celeritas/optical/action/TrackSlotExecutor.hh"

#include "PrimaryGeneratorExecutor.hh"

namespace celeritas
{
namespace detail
{
//---------------------------------------------------------------------------//
/*!
 * Launch a kernel to generate optical photons.
 */
void PrimaryGeneratorAction::generate(optical::CoreParams const& params,
                                      CoreStateDevice& state) const
{
    CELER_EXPECT(state.aux());

    auto const& aux_state = this->counters(*state.aux());
    size_type num_gen
        = min(state.counters().num_vacancies, aux_state.counters.num_pending);

    // Generate optical photons in vacant track slots
    detail::PrimaryGeneratorExecutor execute{
        params.ptr<MemSpace::native>(), state.ptr(), data_, state.counters()};
    static optical::ActionLauncher<decltype(execute)> const launch(*this);
    launch(num_gen, state.stream_id(), execute);
}

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
