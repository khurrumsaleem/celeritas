#include "hip/hip_runtime.h"
//------------------------------ -*- cuda -*- -------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/optical/gen/detail/GeneratorAction.cu
//---------------------------------------------------------------------------//
#include "GeneratorAction.hh"

#include "corecel/Assert.hh"
#include "corecel/sys/KernelLauncher.device.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/optical/CoreParams.hh"
#include "celeritas/optical/CoreState.hh"
#include "celeritas/optical/MaterialParams.hh"
#include "celeritas/optical/action/ActionLauncher.device.hh"
#include "celeritas/optical/action/TrackSlotExecutor.hh"

#include "GeneratorExecutor.hh"
#include "OpticalGenAlgorithms.hh"
#include "UpdateSumExecutor.hh"
#include "../CherenkovGenerator.hh"
#include "../CherenkovParams.hh"
#include "../ScintillationGenerator.hh"
#include "../ScintillationParams.hh"

namespace celeritas
{
namespace detail
{
//---------------------------------------------------------------------------//
/*!
 * Launch a kernel to generate optical photons.
 */
template<GeneratorType G>
void GeneratorAction<G>::generate(optical::CoreParams const& params,
                                  CoreStateDevice& state) const
{
    CELER_EXPECT(state.aux());

    auto& aux_state
        = get<GeneratorState<MemSpace::native>>(*state.aux(), this->aux_id());
    size_type num_gen
        = min(state.counters().num_vacancies, aux_state.counters.num_pending);
    {
        // Generate optical photons in vacant track slots
        detail::GeneratorExecutor<G> execute{params.ptr<MemSpace::native>(),
                                             state.ptr(),
                                             data_.material->device_ref(),
                                             data_.shared->device_ref(),
                                             aux_state.store.ref(),
                                             aux_state.counters.buffer_size,
                                             state.counters()};
        static optical::ActionLauncher<decltype(execute)> const launch(*this);
        launch(num_gen, state.stream_id(), execute);
    }
    {
        // Update the cumulative sum of the number of photons per distribution
        // according to how many were generated
        detail::UpdateSumExecutor execute{aux_state.store.ref(), num_gen};
        static KernelLauncher<decltype(execute)> const launch_kernel(
            "update-sum");
        launch_kernel(
            aux_state.counters.buffer_size, state.stream_id(), execute);
    }
}

//---------------------------------------------------------------------------//
// EXPLICIT INSTANTIATION
//---------------------------------------------------------------------------//

template class GeneratorAction<GeneratorType::cherenkov>;
template class GeneratorAction<GeneratorType::scintillation>;

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
