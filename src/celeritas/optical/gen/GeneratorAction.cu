#include "hip/hip_runtime.h"
//------------------------------ -*- cuda -*- -------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/optical/gen/GeneratorAction.cu
//---------------------------------------------------------------------------//
#include "GeneratorAction.hh"

#include "corecel/Assert.hh"
#include "corecel/sys/KernelLauncher.device.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/optical/CoreParams.hh"
#include "celeritas/optical/CoreState.hh"
#include "celeritas/optical/MaterialParams.hh"
#include "celeritas/optical/action/ActionLauncher.device.hh"
#include "celeritas/optical/action/TrackSlotExecutor.hh"

#include "CherenkovGenerator.hh"
#include "CherenkovParams.hh"
#include "ScintillationGenerator.hh"
#include "ScintillationParams.hh"

#include "detail/GeneratorExecutor.hh"
#include "detail/UpdateSumExecutor.hh"

namespace celeritas
{
namespace optical
{
//---------------------------------------------------------------------------//
/*!
 * Launch a kernel to generate optical photons.
 */
void GeneratorAction::generate(CoreParams const& params,
                               CoreStateDevice& state) const
{
    CELER_EXPECT(state.aux());

    auto& aux_state
        = get<GeneratorState<MemSpace::native>>(*state.aux(), this->aux_id());
    size_type num_gen
        = min(state.counters().num_vacancies, aux_state.counters.num_pending);
    {
        auto cherenkov = data_.cherenkov ? data_.cherenkov->device_ref()
                                         : DeviceCRef<CherenkovData>{};
        auto scint = data_.scintillation ? data_.scintillation->device_ref()
                                         : DeviceCRef<ScintillationData>{};

        // Generate optical photons in vacant track slots
        detail::GeneratorExecutor execute{params.ptr<MemSpace::native>(),
                                          state.ptr(),
                                          data_.material->device_ref(),
                                          cherenkov,
                                          scint,
                                          aux_state.store.ref(),
                                          aux_state.counters.buffer_size,
                                          state.counters()};
        static ActionLauncher<decltype(execute)> const launch(*this);
        launch(num_gen, state.stream_id(), execute);
    }
    {
        // Update the cumulative sum of the number of photons per distribution
        // according to how many were generated
        detail::UpdateSumExecutor execute{aux_state.store.ref(), num_gen};
        static KernelLauncher<decltype(execute)> const launch_kernel(
            "update-sum");
        launch_kernel(
            aux_state.counters.buffer_size, state.stream_id(), execute);
    }
}

//---------------------------------------------------------------------------//
}  // namespace optical
}  // namespace celeritas
