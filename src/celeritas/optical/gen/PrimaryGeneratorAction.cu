#include "hip/hip_runtime.h"
//------------------------------ -*- cuda -*- -------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/optical/gen/PrimaryGeneratorAction.cu
//---------------------------------------------------------------------------//
#include "PrimaryGeneratorAction.hh"

#include "corecel/Assert.hh"
#include "corecel/sys/KernelLauncher.device.hh"
#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/optical/CoreParams.hh"
#include "celeritas/optical/CoreState.hh"
#include "celeritas/optical/MaterialParams.hh"
#include "celeritas/optical/action/ActionLauncher.device.hh"
#include "celeritas/optical/action/TrackSlotExecutor.hh"

#include "detail/PrimaryGeneratorExecutor.hh"

namespace celeritas
{
namespace optical
{
//---------------------------------------------------------------------------//
/*!
 * Launch a kernel to generate optical photons.
 */
void PrimaryGeneratorAction::generate(CoreParams const& params,
                                      CoreStateDevice& state) const
{
    CELER_EXPECT(state.aux());

    auto const& aux_state = this->counters(*state.aux());
    size_type num_gen
        = min(state.counters().num_vacancies, aux_state.counters.num_pending);

    // Generate optical photons in vacant track slots
    detail::PrimaryGeneratorExecutor execute{
        params.ptr<MemSpace::native>(), state.ptr(), data_, state.counters()};
    static ActionLauncher<decltype(execute)> const launch(*this);
    launch(num_gen, state.stream_id(), execute);
}

//---------------------------------------------------------------------------//
}  // namespace optical
}  // namespace celeritas
