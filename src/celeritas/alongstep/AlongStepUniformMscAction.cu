//------------------------------ -*- cuda -*- -------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/alongstep/AlongStepUniformMscAction.cu
//---------------------------------------------------------------------------//
#include "AlongStepUniformMscAction.hh"

#include "corecel/sys/ScopedProfiling.hh"
#include "celeritas/em/params/FluctuationParams.hh"
#include "celeritas/em/params/UrbanMscParams.hh"
#include "celeritas/field/DormandPrinceIntegrator.hh"
#include "celeritas/field/FieldDriverOptions.hh"
#include "celeritas/field/MakeMagFieldPropagator.hh"
#include "celeritas/field/UniformField.hh"
#include "celeritas/global/ActionLauncher.device.hh"
#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"
#include "celeritas/global/TrackExecutor.hh"

#include "detail/AlongStepKernels.hh"
#include "detail/FieldFunctors.hh"
#include "detail/LinearPropagatorFactory.hh"
#include "detail/PropagationApplier.hh"
#include "detail/UniformFieldPropagatorFactory.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
/*!
 * Launch the along-step action on device.
 */
void AlongStepUniformMscAction::step(CoreParams const& params,
                                     CoreStateDevice& state) const
{
    if (this->has_msc())
    {
        detail::launch_limit_msc_step(
            *this, msc_->ref<MemSpace::native>(), params, state);
    }
    auto field = field_->ref<MemSpace::native>();
    {
        ScopedProfiling profile_this{"propagate-uniform"};
        auto execute_thread = ConditionalTrackExecutor{
            params.ptr<MemSpace::native>(),
            state.ptr(),
            detail::IsAlongStepUniformField{this->action_id(), field},
            detail::PropagationApplier{
                detail::UniformFieldPropagatorFactory{field}}};
        static ActionLauncher<decltype(execute_thread)> const launch_kernel(
            *this, "propagate");
        launch_kernel(*this, params, state, execute_thread);
    }
    if (!field_->in_all_volumes())
    {
        // Launch linear propagation kernel for tracks in volumes without field
        ScopedProfiling profile_this{"propagate-linear"};
        auto execute_thread = ConditionalTrackExecutor{
            params.ptr<MemSpace::native>(),
            state.ptr(),
            detail::IsAlongStepLinear{this->action_id(), field},
            detail::PropagationApplier{detail::LinearPropagatorFactory{}}};
        static ActionLauncher<decltype(execute_thread)> const launch_kernel(
            *this, "propagate-linear");
        launch_kernel(*this, params, state, execute_thread);
    }
    if (this->has_msc())
    {
        detail::launch_apply_msc(
            *this, msc_->ref<MemSpace::native>(), params, state);
    }
    detail::launch_update_time(*this, params, state);
    if (this->has_fluct())
    {
        detail::launch_apply_eloss(
            *this, fluct_->ref<MemSpace::native>(), params, state);
    }
    else
    {
        detail::launch_apply_eloss(*this, params, state);
    }
    detail::launch_update_track(*this, params, state);
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
