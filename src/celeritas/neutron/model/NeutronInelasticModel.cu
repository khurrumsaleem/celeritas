//------------------------------ -*- cuda -*- -------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/neutron/model/NeutronInelasticModel.cu
//---------------------------------------------------------------------------//
#include "NeutronInelasticModel.hh"

#include "celeritas/global/CoreParams.hh"
#include "celeritas/global/CoreState.hh"

namespace celeritas
{
//---------------------------------------------------------------------------//
/*!
 * Interact with device data.
 */
void NeutronInelasticModel::step(CoreParams const& params,
                                 CoreStateDevice& state) const
{
    CELER_NOT_IMPLEMENTED("Neutron inelastic interaction");
}

//---------------------------------------------------------------------------//
}  // namespace celeritas
