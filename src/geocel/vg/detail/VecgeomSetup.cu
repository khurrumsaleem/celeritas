//------------------------------ -*- cuda -*- -------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file geocel/vg/detail/VecgeomSetup.cu
//---------------------------------------------------------------------------//
#include "VecgeomSetup.hh"

#include <VecGeom/management/BVHManager.h>

#include "corecel/data/DeviceVector.hh"

#if CELERITAS_VECGEOM_SURFACE
#    include <VecGeom/surfaces/cuda/BrepCudaManager.h>
#endif

#include "corecel/Assert.hh"
#include "corecel/Macros.hh"
#include "corecel/sys/KernelLauncher.device.hh"
#include "corecel/sys/ThreadId.hh"

#if CELERITAS_VECGEOM_SURFACE
using BrepCudaManager = vgbrep::BrepCudaManager<vecgeom::Precision>;
using SurfData = vgbrep::SurfData<vecgeom::Precision>;
#endif

namespace celeritas
{
namespace detail
{
namespace
{
//---------------------------------------------------------------------------//
//! Copy the BVH pointer address to global memory
struct BvhGetter
{
    using pointer_type = CudaBVH_t const*;
    static constexpr char const label[] = "bvh";

    pointer_type* dest{nullptr};

    CELER_FUNCTION void operator()(ThreadId tid)
    {
        CELER_EXPECT(tid == ThreadId{0});
        *dest = vecgeom::cuda::BVHManager::GetBVH(0);
    }
};

//---------------------------------------------------------------------------//
//! Copy the navigation table pointer address to global memory
struct NavIndexGetter
{
    using pointer_type = NavIndex_t const*;
    static constexpr char const label[] = "navindex";

    pointer_type* dest{nullptr};

    __device__ void operator()(ThreadId tid)
    {
        CELER_EXPECT(tid == ThreadId{0});
        *dest = vecgeom::globaldevicegeomdata::gNavIndex;
    }
};

//---------------------------------------------------------------------------//
//! Launch a kernel to copy a value from global memory
template<class GetterT>
auto get_device_pointer()
{
    using pointer_type = typename GetterT::pointer_type;

    // Copy address from inside kernel to GPU global memory
    DeviceVector<pointer_type> temp_global{1, StreamId{}};
    GetterT execute_thread{temp_global.data()};
    static KernelLauncher<decltype(execute_thread)> const launch_kernel(
        "vecgeom-get-" + std::string{GetterT::label});
    launch_kernel(1u, StreamId{}, execute_thread);
    CELER_DEVICE_API_CALL(DeviceSynchronize());

    // Copy address to host
    pointer_type result;
    temp_global.copy_to_host({&result, 1});
    return result;
}

}  // namespace

//---------------------------------------------------------------------------//
/*!
 * Get pointers to the device BVH after setup, for consistency checking.
 */
CudaPointers<CudaBVH_t const> bvh_pointers_device()
{
    CudaPointers<CudaBVH_t const> result;

    // Copy from kernel using 1-thread launch
    result.kernel = get_device_pointer<BvhGetter>();

    // Copy from symbol using runtime API
    CELER_DEVICE_API_CALL(
        MemcpyFromSymbol(&result.symbol,
#if VECGEOM_VERSION >= 0x020000
                         vecgeom::cuda::dBVH<BvhPrecision>,
                         sizeof(vecgeom::cuda::dBVH<BvhPrecision>),
#else
                         vecgeom::cuda::dBVH,
                         sizeof(vecgeom::cuda::dBVH),
#endif
                         0,
                         CELER_DEVICE_API_SYMBOL(MemcpyDeviceToHost)));
    CELER_DEVICE_API_CALL(DeviceSynchronize());

    return result;
}

//---------------------------------------------------------------------------//
/*!
 * Get pointers to the device BVH after setup, for consistency checking.
 */
CudaPointers<NavIndex_t const> navindex_pointers_device()
{
    CudaPointers<NavIndex_t const> result;

    // Copy from kernel using 1-thread launch
    result.kernel = get_device_pointer<NavIndexGetter>();

    // Copy from symbol using runtime API
    CELER_DEVICE_API_CALL(
        MemcpyFromSymbol(&result.symbol,
                         vecgeom::globaldevicegeomdata::gNavIndex,
                         sizeof(vecgeom::globaldevicegeomdata::gNavIndex),
                         0,
                         CELER_DEVICE_API_SYMBOL(MemcpyDeviceToHost)));
    CELER_DEVICE_API_CALL(DeviceSynchronize());

    return result;
}

//---------------------------------------------------------------------------//
// VECGEOM SURFACE
//---------------------------------------------------------------------------//
#if CELERITAS_VECGEOM_SURFACE
void setup_surface_tracking_device(SurfData const& surf_data)
{
    BrepCudaManager::Instance().TransferSurfData(surf_data);
    CELER_DEVICE_API_CALL(DeviceSynchronize());
}

void teardown_surface_tracking_device()
{
    BrepCudaManager::Instance().Cleanup();
}
#endif

//---------------------------------------------------------------------------//
}  // namespace detail
}  // namespace celeritas
