#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright Celeritas contributors: see top-level COPYRIGHT file for details
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file celeritas/field/Fields.test.cu
//---------------------------------------------------------------------------//
#include "Fields.test.hh"

#include <cstdio>

#include "corecel/DeviceRuntimeApi.hh"

#include "corecel/Types.hh"
#include "corecel/cont/Span.hh"
#include "corecel/data/DeviceVector.hh"
#include "corecel/grid/Interpolator.hh"
#include "corecel/math/Algorithms.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "celeritas/field/CartMapField.hh"
#include "celeritas/field/CartMapFieldInput.hh"
#include "celeritas/field/CartMapFieldParams.hh"

#include "TestMacros.hh"

namespace celeritas
{
namespace test
{
namespace
{

using DeviceCRef = CartMapFieldParams::DeviceRef;

//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void field_test_kernel(unsigned int const size,
                                  DeviceCRef field_map_data,
                                  AxisGrid<real_type> x_grid,
                                  AxisGrid<real_type> y_grid,
                                  AxisGrid<real_type> z_grid,
                                  Array<size_type, 3> n_samples,
                                  real_type* field_values)
{
    auto tid = TrackSlotId{KernelParamCalculator::thread_id().unchecked_get()};
    if (tid.get() >= size)
        return;

    CartMapField calc_field(field_map_data);

    size_type nx_samples = static_cast<size_type>(n_samples[0]);
    size_type ny_samples = static_cast<size_type>(n_samples[1]);
    size_type nz_samples = static_cast<size_type>(n_samples[2]);

    size_type index = 0;
    Interpolator interp_x(
        {0, x_grid.min}, {static_cast<real_type>(nx_samples - 1), x_grid.max});
    Interpolator interp_y(
        {0, y_grid.min}, {static_cast<real_type>(ny_samples - 1), y_grid.max});
    Interpolator interp_z(
        {0, z_grid.min}, {static_cast<real_type>(nz_samples - 1), z_grid.max});

    for (size_type ix = 0; ix < nx_samples; ++ix)
    {
        real_type x = interp_x(ix);
        x = celeritas::min(x, x_grid.max - 1);
        for (size_type iy = 0; iy < ny_samples; ++iy)
        {
            real_type y = interp_y(iy);
            y = celeritas::min(y, y_grid.max - 1);
            for (size_type iz = 0; iz < nz_samples; ++iz)
            {
                real_type z = interp_z(iz);
                z = celeritas::min(z, z_grid.max - 1);

                Real3 field = calc_field({x, y, z});
                field_values[index++] = field[0];
                field_values[index++] = field[1];
                field_values[index++] = field[2];
            }
        }
    }
}
}  // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
void field_test(CartMapFieldInput& inp,
                Span<real_type>& field_values,
                Array<size_type, 3>& n_samples)
{
    CartMapFieldParams field_map{inp};

    DeviceVector<real_type> field_values_d(field_values.size());

    DeviceCRef device_cref = field_map.device_ref();

    CELER_LAUNCH_KERNEL(field_test,
                        1,
                        0,
                        1,
                        device_cref,
                        inp.x,
                        inp.y,
                        inp.z,
                        n_samples,
                        field_values_d.data());
    CELER_DEVICE_API_CALL(DeviceSynchronize());

    field_values_d.copy_to_host(field_values);
}

//---------------------------------------------------------------------------//
}  // namespace test
}  // namespace celeritas
