#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2021-2024 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file corecel/math/Algorithms.test.cu
//---------------------------------------------------------------------------//
#include "Algorithms.test.hh"

#include "corecel/DeviceRuntimeApi.hh"

#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"

namespace celeritas
{
namespace test
{
namespace
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void alg_test_kernel(AlgorithmTestData data)
{
    auto tid = KernelParamCalculator::thread_id();

    auto const& inp = data.input;
    auto const& out = data.output;

    if (tid.get() < inp.pi_frac.size())
    {
        sincospi(inp.pi_frac[tid], &out.sinpi[tid], &out.cospi[tid]);
    }
    if (tid.get() < inp.a.size())
    {
        out.fastpow[tid] = fastpow(inp.a[tid], inp.b[tid]);
        out.hypot[tid] = hypot(inp.a[tid], inp.b[tid]);
    }
}
}  // namespace

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device
void alg_test(AlgorithmTestData data)
{
    CELER_LAUNCH_KERNEL(alg_test, data.num_threads, 0, data);
    CELER_DEVICE_API_CALL(DeviceSynchronize());
}

//---------------------------------------------------------------------------//
}  // namespace test
}  // namespace celeritas
